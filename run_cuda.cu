#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "cu_header.h"
#include "cu_helper.h"

__global__ void sum_arrays(int *d_array_a, int *d_array_b,
			   int *d_array_c, int array_length){    
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= array_length) return;
  d_array_c[index] = d_array_a[index] + d_array_b[index];
}

int* run_cuda(int *h_first_array, int *h_second_array, int array_length){
  checkCudaErrors(hipSetDevice(0));
  checkCudaErrors(hipDeviceReset());

  int threads_per_block = 32;

  int ary_byte_size = sizeof(int) * array_length;
  int *d_first_array, *d_second_array, *d_summed_array;
  checkCudaErrors(hipMallocManaged((void**)&d_first_array, ary_byte_size));
  checkCudaErrors(hipMallocManaged((void**)&d_second_array, ary_byte_size));
  checkCudaErrors(hipMallocManaged((void**)&d_summed_array, ary_byte_size));

  checkCudaErrors(hipMemcpy(d_first_array, h_first_array, ary_byte_size,
			     hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_second_array, h_second_array, ary_byte_size,
			     hipMemcpyHostToDevice));

  dim3 blocks(array_length / threads_per_block + 1); //round up just in case
  dim3 threads(threads_per_block);

  clock_t start, stop;
  start = clock();
  
  sum_arrays <<< blocks, threads >>> (d_first_array, d_second_array,
				      d_summed_array, array_length);
  checkCudaErrors(hipDeviceSynchronize());          // synchronize threads
  
  stop = clock();
  double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
  printf("cuda took %f seconds\n", (float)timer_seconds);

  int *h_summed_array = (int *)malloc(sizeof(int) * array_length);
  checkCudaErrors(hipMemcpy(h_summed_array, d_summed_array, ary_byte_size,
			     hipMemcpyDeviceToHost));
		
  checkCudaErrors(hipFree(d_first_array));
  checkCudaErrors(hipFree(d_second_array));
  checkCudaErrors(hipFree(d_summed_array));
  return h_summed_array;
}
